
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>
#include <chrono>
#include <cstdlib> 
#include <vector>

template<typename T>
__global__ void spMulAdd(const int * __restrict__ row, const int * __restrict__ col, const T * __restrict__ val, const T * __restrict__ dx, T * __restrict__ dy, int n, int nnz)
{
    auto tid = threadIdx.x + blockIdx.x * blockDim.x; 
    T y_val = 0.0;

    if (tid < n)
    {
         #pragma unroll
         for (auto j = row[tid]; j < row[tid + 1]; ++j) 
         {
              y_val += val[j] * dx[col[j]];
         }
         dy[tid] = y_val;
         tid += blockIdx.x * blockDim.x;
    }
}

int main(int args, char *argv[])
{
    int n;
    n = atoi(argv[1]);

    int *row, *col; 
    float *val, *vec_x, *vec_y;

    std::unique_ptr<float[]> host_a(new float[n * n]);

    for (auto i = 0; i < n * n; i++)
    {
        if (static_cast<double>(std::rand()) / RAND_MAX < 0.5)
        {
             host_a[i] = static_cast<double>(std::rand()) / RAND_MAX;
        }
        else
        {
             host_a[i] = 0;
        }
    }
    std::unique_ptr<int[]> host_row(new int[n + 1]);
    std::vector<int> host_col;
    std::vector<float> host_val;

    auto nnz = 0;
    host_row[0] = nnz;
    for (auto i = 0; i < n; i++)
    {
        for (auto j = 0; j < n; j++)
        {
            if (host_a[i * n + j] != 0)
            {
                host_val.push_back(host_a[i * n + j]);
                host_col.push_back(j);
                nnz++;
            }
        }
        host_row[i + 1] = nnz;
    }

    std::unique_ptr<float[]> host_x(new float[n]);
    std::unique_ptr<float[]> host_y(new float[n]);

    for (auto i = 0; i < n; i++)
    {
        host_x[i] = static_cast<double>(rand()) / RAND_MAX;
        host_y[i] = 0;
    }

    hipMalloc((void**)&row, (n + 1) * sizeof(int));
    hipMalloc((void**)&col, nnz * sizeof(int));
    hipMalloc((void**)&val, nnz * sizeof(float));
    hipMalloc((void**)&vec_x, n * sizeof(float));
    hipMalloc((void**)&vec_y, n * sizeof(float));

    hipMemcpy(row, host_row.get(), (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    auto* p_host_col = host_col.data();
    hipMemcpy(col, p_host_col, nnz * sizeof(int), hipMemcpyHostToDevice);
    auto* p_host_val = host_val.data();
    hipMemcpy(val, p_host_val, nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vec_x, host_x.get(), n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vec_y, host_y.get(), n * sizeof(float), hipMemcpyHostToDevice);

    auto blocksize = 960;
    dim3 block (blocksize, 1, 1);
    dim3 grid  ((n + blocksize + 1) / block.x, 1, 1);
    
    std::chrono::system_clock::time_point start, end;
    start = std::chrono::system_clock::now();

    spMulAdd<float> <<<grid, block>>>(row, col, val, vec_x, vec_y, n, nnz);

    end = std::chrono::system_clock::now();

    std::unique_ptr<float[]> result(new float[n]);
    hipMemcpy(result.get(), vec_y, n * sizeof(n), hipMemcpyDeviceToHost);

    std::unique_ptr<float[]> host_result(new float[n]);
    for (auto i = 0; i < n; i++)
    {
        host_result[i] = 0;
    }

    for (auto i = 0; i < n; i++)
    {
        for (auto j = 0; j < n; j++)
        {
           host_result[i] += host_a[i * n + j] * host_x[j]; 
        }
    }

    auto checker = 0;
    for (auto i = 0; i < n; i++)
    {
        auto m = 7 - std::log10(n);
        if (fabs(host_result[i] - result[i]) > std::pow(10, -m))
        {
            std::cout << "ng: " << host_result[i] - result[i] << std::endl;
            checker++;
        }
    }
    
    if (checker == 0)
    {
        std::cout << "ok" << std::endl;
    }
    else
    {
        std::cout << checker << std::endl;
    }

    auto time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1000.0);

    std::cout << "n: " << n << ", nnz: " << nnz << ", threads: " << blocksize << std::endl;
    std::cout << "time: " << time << " [ms]" << std::endl;
    std::cout << "perf: " << 2 * n * n / time / 1e6 << " [Gflops/sec]" << std::endl;

    hipFree(row);
    hipFree(col);
    hipFree(val);
    hipFree(vec_x);
    hipFree(vec_y);

    return 0;
}

