
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>
#include <chrono>
#include <cstdlib> 
#include <vector>

//kernel
template<typename T>
__global__ void spMulAdd(const int * __restrict__ row, const int * __restrict__ col, const T * __restrict__ val, const T * __restrict__ dx, T * __restrict__ dy, int n, int nnz)
{
    auto tid = threadIdx.x + blockIdx.x * blockDim.x; 
    T y_val = 0.0;
    
    // Ax=yにおいて、ベクトルyの成分を各スレッドが計算するように並列化
    if (tid < n)
    {
         // C++ は列優先だから、各スレッドは行列Aの各列のデータが読めれば良い
         #pragma unroll
         for (auto j = row[tid]; j < row[tid + 1]; ++j) 
         {
              y_val += val[j] * dx[col[j]];
         }
         dy[tid] = y_val;
         // スレッド番号がnになるまで(yの全要素計算するまで)インクリメント
         tid += blockIdx.x * blockDim.x;
    }
}

template<typename T>
__device__ T warp_reduction(T val)
{
#define warpSize 32

    for (auto offset = warpSize / 2; offset > 0; offset /= 2)
    {
        val += __shfl_down_sync(0xFFFFFFFF, val, offset, warpSize);
    }
    return val;
}

template<typename T>
__global__ void spMulAdd_vector(const int * __restrict__ row, const int * __restrict__ col, const T * __restrict__ val, const T * __restrict__ dx, T * __restrict__ dy, int n, int nnz)
{
    auto tid = threadIdx.x + blockIdx.x * blockDim.x; 
    auto rowid = tid / warpSize;
    auto lane = tid % warpSize;
    T y_val = 0;
    
    if (rowid < n)
    {
         for (auto i = row[rowid] + lane; i < row[rowid + 1]; i += warpSize) 
         {
              y_val += val[i] * dx[col[i]];
         }
         y_val = warp_reduction<T>(y_val);
    }
    

    if (lane == 0 && rowid < n)
    { 
         dy[rowid] = y_val;
    }
}

int main(int args, char *argv[])
{
    // n は実行時引数で与える
    int n;
    n = atoi(argv[1]);

    // 疎行列を作るところ
    int *row, *col; 
    double *val, *vec_x, *vec_y;

    std::unique_ptr<double[]> host_a(new double[n * n]);

    for (auto i = 0; i < n * n; i++)
    {
        if (static_cast<double>(std::rand()) / RAND_MAX < 0.5)
        {
             //host_a[i] = static_cast<double>(std::rand()) / RAND_MAX;
             host_a[i] = 1;
        }
        else
        {
             host_a[i] = 0;
        }
    }
    std::unique_ptr<int[]> host_row(new int[n + 1]);
    std::vector<int> host_col;
    std::vector<double> host_val;

    auto nnz = 0;
    host_row[0] = nnz;
    for (auto i = 0; i < n; i++)
    {
        for (auto j = 0; j < n; j++)
        {
            if (host_a[i * n + j] != 0)
            {
                host_val.push_back(host_a[i * n + j]);
                host_col.push_back(j);
                nnz++;
            }
        }
        host_row[i + 1] = nnz;
    }

    // ベクトルxとベクトルyを作るところ
    std::unique_ptr<double[]> host_x(new double[n]);
    std::unique_ptr<double[]> host_y(new double[n]);

    for (auto i = 0; i < n; i++)
    {
        //host_x[i] = static_cast<double>(rand()) / RAND_MAX;
        host_x[i] = 1;
        host_y[i] = 0;
    }

    // gpu に渡すところ
    hipMalloc((void**)&row, (n + 1) * sizeof(int));
    hipMalloc((void**)&col, nnz * sizeof(int));
    hipMalloc((void**)&val, nnz * sizeof(double));
    hipMalloc((void**)&vec_x, n * sizeof(double));
    hipMalloc((void**)&vec_y, n * sizeof(double));

    hipMemcpy(row, host_row.get(), (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    auto* p_host_col = host_col.data();
    hipMemcpy(col, p_host_col, nnz * sizeof(int), hipMemcpyHostToDevice);
    auto* p_host_val = host_val.data();
    hipMemcpy(val, p_host_val, nnz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vec_x, host_x.get(), n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vec_y, host_y.get(), n * sizeof(double), hipMemcpyHostToDevice);

    // スレッドサイズはどう決めるのがよいのだろうか?
    auto blocksize = 32;
    dim3 block (blocksize, 1, 1);
    dim3 grid  (warpSize * std::ceil(n / static_cast<double>(block.x)), 1, 1);
    
    // 時間計測するところ、データ転送は含まなくてok?
    std::chrono::system_clock::time_point start, end;
    start = std::chrono::system_clock::now();

    // 計算するところ
    spMulAdd_vector<double> <<<grid, block>>>(row, col, val, vec_x, vec_y, n, nnz);

    end = std::chrono::system_clock::now();

    // 結果があっているかcpuでも計算して確認するところ
    std::unique_ptr<double[]> result(new double[n]);
    hipMemcpy(result.get(), vec_y, n * sizeof(n), hipMemcpyDeviceToHost);

    std::unique_ptr<double[]> host_result(new double[n]);
    for (auto i = 0; i < n; i++)
    {
        host_result[i] = 0;
    }

    for (auto i = 0; i < n; i++)
    {
        for (auto j = 0; j < n; j++)
        {
           host_result[i] += host_a[i * n + j] * host_x[j]; 
        }
    }

    auto checker = 0;
    for (auto i = 0; i < n; i++)
    {
        // double で誤差含めてだいたいこのくらい合ってれば正しい？
        auto m = 7 - std::log10(n);
        if (fabs(host_result[i] - result[i]) > std::pow(10, -m))
        {
            // 基準を満たさなかったら NG
            std::cout << "ng: " << result[i] << std::endl;
            checker++;
        }
    }
    
    if (checker == 0)
    {
        std::cout << "ok" << std::endl;
    }
    else
    {
        std::cout << checker << std::endl;
    }

    // 計算時間(データ転送含めない？)や次数、実効性能を出力
    auto time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1000.0);

    std::cout << "n: " << n << ", nnz: " << nnz << ", threads: " << blocksize << std::endl;
    std::cout << "time: " << time << " [ms]" << std::endl;
    std::cout << "perf: " << 2 * n * n / time / 1e6 << " [Gflops/sec]" << std::endl;

    hipFree(row);
    hipFree(col);
    hipFree(val);
    hipFree(vec_x);
    hipFree(vec_y);

    return 0;
}

